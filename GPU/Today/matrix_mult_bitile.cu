
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define TILE_WIDTH 4

__global__
void matMultKernel(float *d_M, float *d_N, float *d_P, int Width){

  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH*2];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH*2];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
 
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH*2 + tx;

  float Pvalue = 0, Pvalue2 = 0;
  int  m,k;
  for(m = 0; m < Width/(TILE_WIDTH*2); ++m){
     Mds[ty][tx] = d_M[Row*Width+m*TILE_WIDTH + tx];
     Mds[ty][tx+TILE_WIDTH] = d_M[Row*Width+m*TILE_WIDTH + tx+TILE_WIDTH];;
     Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty) * Width + Col];
     Nds[ty][tx+TILE_WIDTH] = d_N[(m*TILE_WIDTH + ty)*Width + Col + TILE_WIDTH];
     __syncthreads();
     for(k = 0; k < TILE_WIDTH; ++k){
     	Pvalue += Mds[ty][k] * Nds[k][tx] + Mds[ty][k+TILE_WIDTH] * Nds[k][tx];
	Pvalue2 += Mds[ty][k] * Nds[k][tx+TILE_WIDTH] + Mds[ty][k+TILE_WIDTH] * Nds[k][tx+TILE_WIDTH];
     }
     __syncthreads();
  }
  d_P[Row*Width + Col] = Pvalue;
  d_P[Row*Width + Col + TILE_WIDTH] = Pvalue2;
}

void matMult(float* A, float* B, float* C, int n){
  int size = n*n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  dim3 dimGrid(ceil(n/4.0),ceil(n/4.0),1);
  dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
  matMultKernel<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i,j;
  float *h_A,*h_B,*h_C;
  scanf("%d", &n);
  h_A = (float*) malloc(n*n*sizeof(float));
  h_B = (float*) malloc(n*n*sizeof(float));
  h_C = (float*) malloc(n*n*sizeof(float));
  for(i = 0; i < n; i++){
    //scanf("%f", &h_A[i]);
    for(j = 0; j < n; j++)
      h_A[i*n+j] = 1;
  }
  for(i = 0; i < n; i++){
    //scanf("%f", &h_B[i]);
    for(j = 0; j < n; j++)
      h_B[i*n+j] = 1;
  }
  matMult(h_A,h_B,h_C,n);
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++){
        printf("%f ", h_C[i*n+j]);
    }
    printf("\n");	
  }
  printf("\n");
  return 0;
}
