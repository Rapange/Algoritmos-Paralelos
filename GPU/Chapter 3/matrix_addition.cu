
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void matAddKernel(float *A, float *B, float *C, int n){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if(i < n*n) C[i] = A[i] + B[i];
}

void matAdd(float* A, float* B, float* C, int n){
  int size = n*n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  matAddKernel<<<ceil((n*n)/256.0), 256>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i,j;
  float *h_A,*h_B,*h_C;
  scanf("%d", &n);
  h_A = (float*) malloc(n*n*sizeof(float));
  h_B = (float*) malloc(n*n*sizeof(float));
  h_C = (float*) malloc(n*n*sizeof(float));
  for(i = 0; i < n; i++){
    //scanf("%f", &h_A[i]);
    for(j = 0; j < n; j++)
      h_A[i*n+j] = 1;
  }
  for(i = 0; i < n; i++){
    //scanf("%f", &h_B[i]);
    for(j = 0; j < n; j++)
      h_B[i*n+j] = 1;
  }
  matAdd(h_A,h_B,h_C,n);
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++){
        printf("%f ", h_C[i*n+j]);
    }
    printf("\n");	
  }
  printf("\n");
  return 0;
}
